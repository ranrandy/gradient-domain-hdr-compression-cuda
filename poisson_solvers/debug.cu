#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include "solvers.h"
#include "debug_function2D.h"


int main(int argc, char* argv[]) {
    int method = std::stoi(argv[1]), 
        iterations = std::stoi(argv[2]), 
        checkFrequency = std::stoi(argv[3]), 
        warmup = std::stoi(argv[5]),
        measure = std::stoi(argv[6]); 
    float tolerance = std::stof(argv[4]);
    float omega = (argc == 8) ? std::stof(argv[7]) : 1.45;

    std::cout << std::endl; 
    std::cout << "method: " << method << std::endl; 
    std::cout << "iterations: " << iterations << std::endl; 
    std::cout << "checkFrequency: " << checkFrequency << std::endl; 
    std::cout << "tolerance: " << tolerance << std::endl; 
    std::cout << std::endl; 
    std::cout << "warmup: " << warmup << std::endl; 
    std::cout << "measure: " << measure << std::endl;
    std::cout << "omega: " << omega << std::endl;
    std::cout << std::endl; 
    
    // --------------------------------------------- 2D ---------------------------------------------
    int W = 1600, H = 1200;
    Function2D function2d(W, H);

    int j = 9, i = 25;
    float real_laplacian = function2d._real_f_laplacian[j * W + i];
    float estimate_laplacian = function2d._estimated_f_laplacian[j * W + i];

    std::cout << "Function: f(x) = sin(pi*(x+y)/100.0): " << std::endl;
    std::cout << "L1 Estimated Laplacian Error " << std::setprecision (6) << fabs(estimate_laplacian - real_laplacian) << std::endl;
    std::cout << std::endl;

    // ---------------------------------- Solve Poisson Equation ------------------------------------
    float *d_divG;
    hipMalloc(&d_divG, H * W * sizeof(float));
    hipMemcpy(d_divG, function2d._estimated_f_laplacian, H * W * sizeof(float), hipMemcpyHostToDevice);

    float *d_I_log;
    hipMalloc(&d_I_log, H * W * sizeof(float));

    float *h_I_log;
    hipHostMalloc(&h_I_log, H * W * sizeof(float));

    int iter_converge = 0;

    float *args;
    hipHostMalloc(&args, sizeof(float));
    args[0] = omega;

    // Warm up
    for (int iter = 0; iter < warmup; ++iter)
    {
        iter_converge = simpleSolver(H, W, d_divG, method, args, iterations, tolerance, checkFrequency, d_I_log);
        hipDeviceSynchronize();

        hipMemcpy(h_I_log, d_I_log, H * W * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }

    // Set up timer
    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timer
    hipEventRecord(start);

    for (int iter = 0; iter < measure; ++iter)
    {
        iter_converge = simpleSolver(H, W, d_divG, method, args, iterations, tolerance, checkFrequency, d_I_log);
        hipDeviceSynchronize();

        hipMemcpy(h_I_log, d_I_log, H * W * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }

    // Stop timer
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time duration: " << std::setprecision (6) << milliseconds / measure << " milliseconds" << std::endl;
    std::cout << std::endl;

    // --------------------------------------- Calculate Error --------------------------------------
    float error = 0.0;
    for (int i = 0; i < H * W; ++i) {
        error += fabs(h_I_log[i] - function2d._f[i]);
    }
    error /= H * W;
    std::cout << "Error: " << std::setprecision (6) << error << std::endl;
    std::cout << "Number of iteration until convergence: " << std::setprecision (6) << iter_converge << std::endl;
    std::cout << std::endl;

    // --------------------------------------- Free Memory --------------------------------------
    hipFree(d_divG);
    hipFree(d_I_log);
    hipHostFree(h_I_log);
    return 0;
}