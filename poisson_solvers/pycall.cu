#include "pycall.h"
#include "solvers.h"

int solve(
    const int H, const int W,
    const torch::Tensor& d_div_G, 
    const int method, const std::vector<float>& args,
    const int iterations, const int checkFrequency, const float tolerance,
    torch::Tensor& h_I_log)
{
    float* arguments;

    float *d_I_log;
    hipMalloc(&d_I_log, H * W * sizeof(float));
    hipMemset(d_I_log, 0.0, H * W * sizeof(float));

    int iter_converge = 0;

    std::cout << "method: " << method << std::endl;

    // Set up timer
    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Start timer
    hipEventRecord(start);
    
    if (method <= 4) // Simple Solver
    {
        // std::cout << "iterations: " << iterations << std::endl; 
        // std::cout << "checkFrequency: " << checkFrequency << std::endl; 
        // std::cout << "tolerance: " << tolerance << std::endl; 

        assert (args.size() == 0 || args.size() == 1);

        hipHostMalloc(&arguments, 1 * sizeof(float));
        arguments[0] = (args.size() == 1) ? args[0] : 1.90;

        // std::cout << "omega: " << arguments[0] << std::endl;
        
        iter_converge = simpleSolver(
            H, W, d_div_G.contiguous().data<float>(),
            method, arguments,
            iterations, checkFrequency, tolerance,
            d_I_log);
    }
    else // Multigrid Solver
    {
        // std::cout << "cycleIterations: " << iterations << std::endl; 
        // std::cout << "checkCycleFrequency: " << checkFrequency << std::endl; 
        // std::cout << "cycleTolerance: " << tolerance << std::endl; 

        assert (args.size() == 6 || args.size() == 7);

        hipHostMalloc(&arguments, 7 * sizeof(float));
        for (int i = 0; i < args.size(); ++i) arguments[i] = args[i];
        arguments[6] = (args.size() == 7) ? args[0] : 1.90;

        // std::cout << "multigridSmoothingMethod: " << arguments[0] << std::endl; 
        // std::cout << "prepostSmoothingIterations: " << arguments[1] << std::endl; 
        // std::cout << std::endl; 
        // std::cout << "coarsestSideLength: " << arguments[2] << std::endl; 
        // std::cout << std::endl; 
        // std::cout << "multigridCoarsestIterations: " << arguments[3] << std::endl; 
        // std::cout << "checkCoarsestFrequency: " << arguments[4] << std::endl; 
        // std::cout << "CoarsestTolerance: " << arguments[5] << std::endl; 
        // std::cout << std::endl; 
        // std::cout << "omega: " << arguments[6] << std::endl;

        iter_converge = multigridSolver(
            H, W, d_div_G.contiguous().data<float>(),
            method, arguments,
            iterations, checkFrequency, tolerance,
            d_I_log);
    }
    hipDeviceSynchronize();

    hipMemcpy(h_I_log.contiguous().data<float>(), d_I_log, H * W * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Stop timer
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time duration: " << std::setprecision (6) << milliseconds << " milliseconds" << std::endl;
    std::cout << std::endl;

    hipFree(d_I_log);
    hipHostFree(arguments);

    return iter_converge;
}