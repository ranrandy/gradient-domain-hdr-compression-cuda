#include "hip/hip_runtime.h"
#include "solvers.h"


/*
    Kernels for computing the residual/error between the previous and current iteration results.
    In most cases, atomicAdd should be good.
*/
__global__ void atomicAddBlockErrorsKernel(const float* current, const float* previous, float* result, const int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float s_sum[1024 / WARP_SIZE];

    int nwarps = blockDim.x / WARP_SIZE;
    int my_warp = threadIdx.x / WARP_SIZE;

    float sum = 0.0;
    
    if (tid < N)
        sum = fabsf(current[tid] - previous[tid]);
    __syncwarp();

    // shift within warp
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) 
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
    
    // sum over warps if needed
    if (nwarps > 1) {
        if (threadIdx.x % WARP_SIZE == 0)
            s_sum[my_warp] = sum;
        __syncthreads();

        if (threadIdx.x == 0) {
            for (int i = 1; i < nwarps; ++i)
                sum += s_sum[i];
        }
    }

    // final step - store results into the main device memory
    if (threadIdx.x == 0) {
        atomicAdd(result, sum);
    }
}

/*
    Kernel for the Jacobi method.
*/
__global__ void jacobiKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    float up = (j > 0) ? current[idx - W] : current[idx];
    float left = (i > 0) ? current[idx - 1] : current[idx];
    float right = (i < W - 1) ? current[idx + 1] : current[idx];
    float bottom = (j < H - 1) ? current[idx + W] : current[idx];

    result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
}

void jacobi(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_I_log)
{
    jacobiKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_I_log);
}


/*
    Kernels for the Red-Black Gauss-Seildel method.
*/
__global__ void redGaussSeidelKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float up = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel(const int H, const int W, const float* b, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float up = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];
        result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
}

void gaussSeidelRedBlack(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_I_log)
{
    redGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_I_log);
    blackGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_I_log);
}


/*
    Kernels for the Red-Black Gauss-Seildel method with pre-reordered grids.
*/
__global__ void redGaussSeidelKernel2(const int H, const int W, const float* b, const float *black, float *red) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;
        int b_idx = orig_j * W + i;

        float up = black[idx];
        float left = black[idx - 1];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (j < H - 1) ? black[idx + W] : red[idx];
        red[idx] = 0.25f * (up + left + right + bottom - b[b_idx]);
    }
    else
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;
        int b_idx = orig_j * W + i;

        float up = (j > 0) ? black[idx - W] : red[idx];
        float left = (i > 0) ? black[idx - 1] : red[idx];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (j < H - 1) ? black[idx] : red[idx];
        red[idx] = 0.25f * (up + left + right + bottom - b[b_idx]);
    }
}

__global__ void blackGaussSeidelKernel2(const int H, const int W, const float* b, const float *red, float *black) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;
        int b_idx = orig_j * W + i;

        float up = (orig_j > 0) ? red[idx - W] : black[idx];
        float left = red[idx - 1];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx] : black[idx];
        black[idx] = 0.25f * (up + left + right + bottom - b[b_idx]);
    }
    else
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;
        int b_idx = orig_j * W + i;

        float up = red[idx];
        float left = (i > 0) ? red[idx - 1] : black[idx];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx + W] : black[idx];
        black[idx] = 0.25f * (up + left + right + bottom - b[b_idx]);
    }
}

__global__ void fillInGaussSeidel2Kernel(const int H, const int W, const float* red, const float* black, float* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;
    int squeezed_j;

    if ((j + i) % 2 == 0) // red
    {
        if (i % 2 == 1) squeezed_j = (j - 1) / 2;
        else squeezed_j = j / 2;
        int squeezed_idx = squeezed_j * W + i; 
        result[idx] = red[squeezed_idx];
    }
    else // black
    {
        if (i % 2 == 1) squeezed_j = j / 2;
        else squeezed_j = (j - 1) / 2; 
        int squeezed_idx = squeezed_j * W + i; 
        result[idx] = black[squeezed_idx];
    }
}

void gaussSeidelRedBlack2(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* red, float* black)
{
    redGaussSeidelKernel2<<<nblocks, nthreads>>>(H, W, d_divG, black, red);
    blackGaussSeidelKernel2<<<nblocks, nthreads>>>(H, W, d_divG, red, black);
}


/*
    Kernels for the Red-Black Gauss-Seildel method with overrelaxation.
*/
__global__ void redGaussSeidelSORKernel(const int H, const int W, const float* b, const float *current, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float up = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        
        float GS_update = 0.25f * (up + left + right + bottom - b[idx]);
        result[idx] = (1 - w_opt) * current[idx] + w_opt * GS_update;
    }
}

__global__ void blackGaussSeidelSORKernel(const int H, const int W, const float* b, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float up = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];

        float GS_update = 0.25f * (up + left + right + bottom - b[idx]);
        result[idx] = (1 - w_opt) * result[idx] + w_opt * GS_update;
    }
}

void gaussSeidelRedBlackSOR(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_I_log)
{
    float w_opt = 2.0f / (1.0f + sqrtf(1.0f - powf(cosf(M_PI / max(H, W)), 2.0f)));
    redGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_I_log, w_opt);
    blackGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_I_log, w_opt);
}


int solver(
    const int H, const int W, 
    const float* d_divG, const int method,
    const int iterations, const float tolerance, const int checkFrequency,
    float* d_I_log)
{
    const int N = H * W;

    methodFunction methodKernel;
    switch (method) {
        case 0:
            methodKernel = jacobi;
            break;
        case 1:
            methodKernel = gaussSeidelRedBlack;
            break;
        case 2:
            methodKernel = gaussSeidelRedBlack2;
            break;
        case 3:
            methodKernel = gaussSeidelRedBlackSOR;
            break;
        default:
            return -1;
    }

    int nblocksError = (N + MAX_THREADS - 1) / MAX_THREADS;
    float error_h;
    float *error_d;
    hipMalloc(&error_d, sizeof(float));

    if (method == 0 || method == 1) 
    {                
        dim3 nthreadsMethod(16, 16, 1);
        dim3 nblocksMethod((W + nthreadsMethod.x - 1) / nthreadsMethod.x, (H + nthreadsMethod.y - 1) / nthreadsMethod.y, 1);

        float *d_current, *d_result;
        hipMalloc(&d_current, N * sizeof(float));
        hipMemset(d_current, 0.0, N * sizeof(float));
        hipMalloc(&d_result, N * sizeof(float));
        hipMemset(d_result, 0.0, N * sizeof(float));

        int i = 0;
        for (; i < iterations; ) 
        {
            methodKernel(H, W, d_divG, nblocksMethod, nthreadsMethod, d_current, d_result);
            hipDeviceSynchronize(); 
            std::swap(d_current, d_result); ++i;

            if (i % checkFrequency == 0)
            {   
                hipMemset(error_d, 0.0, sizeof(float));
                atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(d_current, d_result, error_d, N);
                hipDeviceSynchronize();
                hipMemcpy(&error_h, error_d, sizeof(float), hipMemcpyDeviceToHost);
                error_h /= N;

                if (error_h < tolerance) break;
            }
        }

        if (i % 2 == 1) std::swap(d_current, d_result);
        hipMemcpy(d_I_log, d_result, N * sizeof(float), hipMemcpyDeviceToDevice);
        hipFree(d_current);
        hipFree(d_result);
        hipFree(error_d);
        return i;
    }

    if (method == 2) 
    {
        dim3 nthreadsMethod(32, 16, 1);
        dim3 nblocksMethod((W + nthreadsMethod.x - 1) / nthreadsMethod.x, (int(H/2) + nthreadsMethod.y - 1) / nthreadsMethod.y, 1);

        float *red, *black;
        hipMalloc(&red, int(N / 2) * sizeof(float));
        hipMemset(red, 0.0, int(N / 2) * sizeof(float));
        hipMalloc(&black, int(N / 2) * sizeof(float));
        hipMemset(black, 0.0, int(N / 2) * sizeof(float));

        float *prev_red;
        hipMalloc(&prev_red, int(N / 2) * sizeof(float));

        int i = 0;
        for (; i < iterations; ) 
        {
            if ((i+1) % checkFrequency == 0) hipMemcpy(prev_red, red, int(N / 2) * sizeof(float), hipMemcpyDeviceToDevice);

            methodKernel(H, W, d_divG, nblocksMethod, nthreadsMethod, red, black);
            hipDeviceSynchronize();
            ++i;

            if (i % checkFrequency == 0)
            {   
                hipMemset(error_d, 0.0, sizeof(float));
                atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(red, prev_red, error_d, N / 2);
                hipDeviceSynchronize();
                hipMemcpy(&error_h, error_d, sizeof(float), hipMemcpyDeviceToHost);
                error_h /= N / 2;

                if (error_h < tolerance) break;
            }
        }

        dim3 nthreadsFillIn(16, 16, 1);
        dim3 nblocksFillIn((W + nthreadsFillIn.x - 1) / nthreadsFillIn.x, (H + nthreadsFillIn.y - 1) / nthreadsFillIn.y, 1);
        fillInGaussSeidel2Kernel<<<nblocksFillIn, nthreadsFillIn>>>(H, W, red, black, d_I_log);
        hipDeviceSynchronize();

        hipFree(red);
        hipFree(black);
        hipFree(prev_red);
        hipFree(error_d);
        return i;
    }

    hipFree(error_d);
    return -1;
}


void fullMultigridSolver(    
    const int H, const int W, 
    const float* d_divG, 
    const int iterations, const float tolerence,
    float* d_I_log)
{

}
