#include "hip/hip_runtime.h"
#include "solvers.h"


/*
    Computing the error between the previous and current iteration results. In most cases, atomicAdd should be fast enough.
*/
__global__ void atomicAddBlockErrorsKernel(const float* current, const float* previous, float* result, const int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float s_sum[1024 / WARP_SIZE];

    int nwarps = blockDim.x / WARP_SIZE;
    int my_warp = threadIdx.x / WARP_SIZE;

    float sum = 0.0;
    
    if (tid < N)
        sum = fabsf(current[tid] - previous[tid]);
    __syncwarp();

    // shift within warp
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) 
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
    
    // sum over warps if needed
    if (nwarps > 1) {
        if (threadIdx.x % WARP_SIZE == 0)
            s_sum[my_warp] = sum;
        __syncthreads();

        if (threadIdx.x == 0) {
            for (int i = 1; i < nwarps; ++i)
                sum += s_sum[i];
        }
    }

    // final step - store results into the main device memory
    if (threadIdx.x == 0) {
        atomicAdd(result, sum);
    }
}

/*
    0. Jacobi method.
*/
__global__ void jacobiKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    float top = (j > 0) ? current[idx - W] : current[idx];
    float left = (i > 0) ? current[idx - 1] : current[idx];
    float right = (i < W - 1) ? current[idx + 1] : current[idx];
    float bottom = (j < H - 1) ? current[idx + W] : current[idx];

    result[idx] = 0.25f * (top + left + right + bottom - b[idx]);
}

void jacobi(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_result, const float* args)
{
    jacobiKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result);
}


/*
    1. Red-Black Gauss-Seildel method.
*/
__global__ void redGaussSeidelKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float top = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        result[idx] = 0.25f * (top + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel(const int H, const int W, const float* b, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float top = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];
        result[idx] = 0.25f * (top + left + right + bottom - b[idx]);
    }
}

void gaussSeidelRedBlack(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_result, const float* args)
{
    redGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result);
    blackGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_result);
}


/*
    2. Red-Black Gauss-Seildel method with successive over relaxation (SOR).
*/
__global__ void redGaussSeidelSORKernel(const int H, const int W, const float* b, const float *current, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float top = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        result[idx] = (1 - w_opt) * current[idx] + w_opt * 0.25f * (top + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelSORKernel(const int H, const int W, const float* b, const float *current, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float top = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];
        result[idx] = (1 - w_opt) * current[idx] + w_opt * 0.25f * (top + left + right + bottom - b[idx]);
    }
}

void gaussSeidelRedBlackSOR(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_result, const float* args)
{
    // float w_opt = 2.0f / (1.0f + sinf(M_PI / max(H, W)));
    redGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result, args[0]);
    blackGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result, args[0]);
}


/*
    --> Red-Black pre-reordering of grids.
*/
__global__ void fillInRedBlackInitGaussSeidel2Kernel(const int H, const int W, const float* init_guess, float* red, float* black) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;
    int squeezed_j;

    if ((i+j) % 2 == 0) // red
    {
        if (i % 2 == 1) squeezed_j = (j - 1) / 2;
        else squeezed_j = j / 2;
        red[squeezed_j * W + i] = init_guess[idx];
    }
    else // black
    {
        if (i % 2 == 1) squeezed_j = j / 2;
        else squeezed_j = (j - 1) / 2; 
        black[squeezed_j * W + i] = init_guess[idx];
    }
}

__global__ void fillInRedBlackBGaussSeidel2Kernel(const int H, const int W, const float* b, float* red_black_b, const int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;
    int squeezed_j;

    if ((i+j) % 2 == 0) // red
    {
        if (i % 2 == 1) squeezed_j = (j - 1) / 2;
        else squeezed_j = j / 2;
        red_black_b[squeezed_j * W + i] = b[idx];
    }
    else // black
    {
        if (i % 2 == 1) squeezed_j = j / 2;
        else squeezed_j = (j - 1) / 2; 
        red_black_b[squeezed_j * W + i + offset] = b[idx];
    }
}

__global__ void fillInGaussSeidel2Kernel(const int H, const int W, const float* red, const float* black, float* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;
    int squeezed_j;

    if ((j + i) % 2 == 0) // red
    {
        if (i % 2 == 1) squeezed_j = (j - 1) / 2;
        else squeezed_j = j / 2;
        int squeezed_idx = squeezed_j * W + i; 
        result[idx] = red[squeezed_idx];
    }
    else // black
    {
        if (i % 2 == 1) squeezed_j = j / 2;
        else squeezed_j = (j - 1) / 2; 
        int squeezed_idx = squeezed_j * W + i; 
        result[idx] = black[squeezed_idx];
    }
}

/*
    3. Gauss-Seidel with pre-reordering of grids.
*/
__global__ void redGaussSeidelKernel2(const int H, const int W, const float* b, const float *black, float *red) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float top = black[idx];
        float left = black[idx - 1];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx + W] : red[idx];
        red[idx] = 0.25f * (top + left + right + bottom - b[idx]);
    }
    else
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float top = (orig_j > 0) ? black[idx - W] : red[idx];
        float left = (i > 0) ? black[idx - 1] : red[idx];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx] : red[idx];
        red[idx] = 0.25f * (top + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel2(const int H, const int W, const float* b, const float *red, float *black, const int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float top = (orig_j > 0) ? red[idx - W] : black[idx];
        float left = red[idx - 1];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx] : black[idx];
        black[idx] = 0.25f * (top + left + right + bottom - b[idx + offset]);
    }
    else
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float top = red[idx];
        float left = (i > 0) ? red[idx - 1] : black[idx];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx + W] : black[idx];
        black[idx] = 0.25f * (top + left + right + bottom - b[idx + offset]);
    }
}

void gaussSeidelRedBlack2(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* red, float* black, const float* args)
{
    redGaussSeidelKernel2<<<nblocks, nthreads>>>(H, W, d_divG, black, red);
    blackGaussSeidelKernel2<<<nblocks, nthreads>>>(H, W, d_divG, red, black, (H * W) / 2);
}

/*
    4. Gauss-Seidel with SOR and pre-reordering of grids.
*/
__global__ void redGaussSeidelKernel2SOR(const int H, const int W, const float* b, const float *black, float *red, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float top = black[idx];
        float left = black[idx - 1];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx + W] : red[idx];
        red[idx] = (1 - w_opt) * red[idx] + w_opt * 0.25f * (top + left + right + bottom - b[idx]);
    }
    else
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float top = (orig_j > 0) ? black[idx - W] : red[idx];
        float left = (i > 0) ? black[idx - 1] : red[idx];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx] : red[idx];
        red[idx] = (1 - w_opt) * red[idx] + w_opt * 0.25f * (top + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel2SOR(const int H, const int W, const float* b, const float *red, float *black, const int offset, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float top = (orig_j > 0) ? red[idx - W] : black[idx];
        float left = red[idx - 1];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx] : black[idx];
        black[idx] = (1 - w_opt) * black[idx] + w_opt * 0.25f * (top + left + right + bottom - b[idx + offset]);
    }
    else
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float top = red[idx];
        float left = (i > 0) ? red[idx - 1] : black[idx];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx + W] : black[idx];
        black[idx] = (1 - w_opt) * black[idx] + w_opt * 0.25f * (top + left + right + bottom - b[idx + offset]);
    }
}

void gaussSeidelRedBlack2SOR(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* red, float* black, const float* args)
{
    redGaussSeidelKernel2SOR<<<nblocks, nthreads>>>(H, W, d_divG, black, red, args[0]);
    blackGaussSeidelKernel2SOR<<<nblocks, nthreads>>>(H, W, d_divG, red, black, (H * W) / 2, args[0]);
}


int simpleSolver(
    const int H, const int W, 
    const float* d_divG, const int method, const float* args, 
    const int iterations, const int checkFrequency, const float tolerance,
    float* d_I_log)
{
    const int N = H * W, N2 = H * W / 2;

    methodFunction methodKernel;
    switch (method) {
        case 0:
            methodKernel = jacobi;
            break;
        case 1:
            methodKernel = gaussSeidelRedBlack;
            break;
        case 2:
            methodKernel = gaussSeidelRedBlackSOR;
            break;
        case 3:
            methodKernel = gaussSeidelRedBlack2;
            break;
        case 4:
            methodKernel = gaussSeidelRedBlack2SOR;
            break;
        default:
            return -1;
    }

    int nblocksError = (N + MAX_THREADS - 1) / MAX_THREADS;
    float h_error;
    float *d_error;
    hipMalloc(&d_error, sizeof(float));

    if (method <= 2) 
    {                
        dim3 nthreadsMethod(16, 16, 1);
        dim3 nblocksMethod((W + nthreadsMethod.x - 1) / nthreadsMethod.x, (H + nthreadsMethod.y - 1) / nthreadsMethod.y, 1);

        float *d_current, *d_result;
        hipMalloc(&d_current, N * sizeof(float));
        hipMemset(d_current, 0.0, N * sizeof(float));
        if (d_I_log != nullptr) hipMemcpy(d_current, d_I_log, N * sizeof(float), hipMemcpyDeviceToDevice);

        hipMalloc(&d_result, N * sizeof(float));
        hipMemset(d_result, 0.0, N * sizeof(float));

        int i = 0;
        for (; i < iterations; ) 
        {
            methodKernel(H, W, d_divG, nblocksMethod, nthreadsMethod, d_current, d_result, args);
            hipDeviceSynchronize(); 
            std::swap(d_current, d_result); ++i;

            if (i % checkFrequency == 0)
            {   
                hipMemset(d_error, 0.0, sizeof(float));
                atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(d_current, d_result, d_error, N);
                hipDeviceSynchronize();
                hipMemcpy(&h_error, d_error, sizeof(float), hipMemcpyDeviceToHost);
                h_error /= N;

                if (h_error < tolerance) break;
            }
        }

        if (i % 2 == 1) std::swap(d_current, d_result);
        hipMemcpy(d_I_log, d_result, N * sizeof(float), hipMemcpyDeviceToDevice);
        hipFree(d_current);
        hipFree(d_result);
        hipFree(d_error);
        return i;
    }

    if (method > 2) 
    {
        dim3 nthreadsMethod(32, 16, 1);
        dim3 nblocksMethod((W + nthreadsMethod.x - 1) / nthreadsMethod.x, (int(H/2) + nthreadsMethod.y - 1) / nthreadsMethod.y, 1);

        dim3 nthreadsFillIn(16, 16, 1);
        dim3 nblocksFillIn((W + nthreadsFillIn.x - 1) / nthreadsFillIn.x, (H + nthreadsFillIn.y - 1) / nthreadsFillIn.y, 1);

        float *red, *black;
        hipMalloc(&red, N2 * sizeof(float));
        hipMemset(red, 0.0, N2 * sizeof(float));
        hipMalloc(&black, N2 * sizeof(float));
        hipMemset(black, 0.0, N2 * sizeof(float));
        if (d_I_log != nullptr) fillInRedBlackInitGaussSeidel2Kernel<<<nblocksFillIn, nthreadsFillIn>>>(H, W, d_I_log, red, black);

        float *prev_red;
        hipMalloc(&prev_red, N2 * sizeof(float));

        float *red_black_divG;
        hipMalloc(&red_black_divG, N * sizeof(float));
        fillInRedBlackBGaussSeidel2Kernel<<<nblocksFillIn, nthreadsFillIn>>>(H, W, d_divG, red_black_divG, N2);

        int i = 0;
        for (; i < iterations; ) 
        {
            if ((i+1) % checkFrequency == 0) hipMemcpy(prev_red, red, N2 * sizeof(float), hipMemcpyDeviceToDevice);

            methodKernel(H, W, red_black_divG, nblocksMethod, nthreadsMethod, red, black, args);
            hipDeviceSynchronize();
            ++i;

            if (i % checkFrequency == 0)
            {   
                hipMemset(d_error, 0.0, sizeof(float));
                atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(red, prev_red, d_error, N2);
                hipDeviceSynchronize();
                hipMemcpy(&h_error, d_error, sizeof(float), hipMemcpyDeviceToHost);
                h_error /= N2;

                if (h_error < tolerance) break;
            }
        }

        fillInGaussSeidel2Kernel<<<nblocksFillIn, nthreadsFillIn>>>(H, W, red, black, d_I_log);
        hipDeviceSynchronize();

        hipFree(red);
        hipFree(black);
        hipFree(prev_red);
        hipFree(red_black_divG);
        hipFree(d_error);
        return i;
    }

    hipFree(d_error);
    return -1;
}


__global__ void computeResidualKernel(const int H, const int W, const float* b_h, const float* u_h, float* r_h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    float top = (j > 0) ? u_h[idx - W] : u_h[idx];
    float left = (i > 0) ? u_h[idx - 1] : u_h[idx];
    float right = (i < W - 1) ? u_h[idx + 1] : u_h[idx];
    float bottom = (j < H - 1) ? u_h[idx + W] : u_h[idx];

    r_h[idx] = b_h[idx] - (top + left + right + bottom - 4 * u_h[idx]);
}

__global__ void restrict2DKernel(const int H, const int W, const int H2, const int W2, const float* r_h, float* r_2h) {
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    int j2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (i2 >= W2 || j2 >= H2) return;

    int i = i2 * 2;
    int j = j2 * 2;

    int idx = j * W + i;
    int idx2 = j2 * W2 + i2;

    float top = (j > 0) ? r_h[idx - W] : r_h[idx];
    float left = (i > 0) ? r_h[idx - 1] : r_h[idx];
    float right = (i < W - 1) ? r_h[idx + 1] : r_h[idx];
    float bottom = (j < H - 1) ? r_h[idx + W] : r_h[idx];

    float topLeft = (j > 0) ? ((i > 0) ? r_h[idx-W-1] : r_h[idx-W]) : ((i > 0) ? r_h[idx-1] : r_h[idx]);
    float topRight = (j > 0) ? ((i < W - 1) ? r_h[idx-W+1] : r_h[idx-W]) : ((i < W - 1) ? r_h[idx+1] : r_h[idx]);
    float bottomLeft = (j < H - 1) ? ((i > 0) ? r_h[idx+W-1] : r_h[idx+W]) : ((i > 0) ? r_h[idx-1] : r_h[idx]);
    float bottomRight = (j < H - 1) ? ((i < W - 1) ? r_h[idx+W+1] : r_h[idx+W]) : ((i < W - 1) ? r_h[idx+1] : r_h[idx]);

    r_2h[idx2] = 0.0625 * (topLeft + topRight + bottomLeft + bottomRight) + 0.125 * (top + bottom + left + right) + 0.25 * r_h[idx];
}

__global__ void interpolate2DKernel(const int H, const int W, const int W2, const float* E_2h, float* E_h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    int i2 = i / 2;
    int j2 = j / 2;
    int idx2 = j2 * W2 + i2;
    
    if (j % 2 == 0)
    {
        if (i % 2 == 0)
        {
            E_h[idx] = E_2h[idx2];
        }
        else
        {
            E_h[idx] = 0.5 * (E_2h[idx2] + E_2h[idx2 + 1]);
        }
    }
    else
    {
        if (i % 2 == 0)
        {
            E_h[idx] = 0.5 * (E_2h[idx2] + E_2h[idx2 + W2]);
        }
        else
        {
            E_h[idx] = 0.5 * (E_2h[idx2] + E_2h[idx2 + 1] + E_2h[idx2 + W2] + E_2h[idx2 + W2 + 1]);
        }
    }
}

__global__ void add2DKernel(const int H, const int W, const float* E_h, float* u_h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    u_h[idx] += E_h[idx];
}

int vCycleSolver(    
    const int H, const int W, 
    const float* d_divG, const float* args,
    float* d_I_log)
{
    const int N = H * W;
    const int H2 = std::ceil(H / 2.0), W2 = std::ceil(W / 2.0);
    const int N2 = H2 * W2;

    dim3 nthreads_h(16, 16, 1);
    dim3 nblocks_h((W + nthreads_h.x - 1) / nthreads_h.x, (H + nthreads_h.y - 1) / nthreads_h.y, 1);

    dim3 nthreads_2h(16, 16, 1);
    dim3 nblocks_2h((W2 + nthreads_2h.x - 1) / nthreads_2h.x, (H2 + nthreads_2h.y - 1) / nthreads_2h.y, 1);
    
    // Step 1: Iterate on A_h * u = b_h to reach u_h (say 3 Jacobi or Gauss-Seidel steps)
    int pre_smoothing_iter = simpleSolver(H, W, d_divG, args[0], args+6, args[1], args[4], args[5], d_I_log);
    hipDeviceSynchronize();

    // Step 2: Restrict the residual r_h = b_h − A_h * u_h to the coarse grid by r_{2h} = R_{h}^{2h} * r_h
    float *d_r_h;
    hipMalloc(&d_r_h, N * sizeof(float));
    computeResidualKernel<<<nblocks_h, nthreads_h>>>(H, W, d_divG, d_I_log, d_r_h);
    hipDeviceSynchronize();

    float *d_r_2h;
    hipMalloc(&d_r_2h, N2 * sizeof(float));
    restrict2DKernel<<<nblocks_2h, nthreads_2h>>>(H, W, H2, W2, d_r_h, d_r_2h);
    hipDeviceSynchronize();

    // Step 3: Solve A_{2h} * E_{2h} = r_{2h} (or come close to E_{2h} by 3 iterations from E = 0)
    float *d_E_2h;
    hipMalloc(&d_E_2h, N2 * sizeof(float));
    hipMemset(d_E_2h, 0.0, N2 * sizeof(float));
    int cycle_smoothing_iter;
    if (std::min(H2, W2) <= args[2])
        cycle_smoothing_iter = simpleSolver(H2, W2, d_r_2h, args[0], args+6, args[3], args[4], args[5], d_E_2h);
    else
        cycle_smoothing_iter = vCycleSolver(H2, W2, d_r_2h, args, d_E_2h);
    hipDeviceSynchronize();

    // Step 4: Interpolate E_{2h} back to E_h = I_{2h}^h * E_{2h}. Add E_h to u_h
    float *d_E_h;
    hipMalloc(&d_E_h, N * sizeof(float));
    interpolate2DKernel<<<nblocks_h, nthreads_h>>>(H, W, W2, d_E_2h, d_E_h);
    hipDeviceSynchronize();

    add2DKernel<<<nblocks_h, nthreads_h>>>(H, W, d_E_h, d_I_log);
    hipDeviceSynchronize();

    // Step 5: Iterate 3 more times on A_h * u = b_h starting from the improved u_h + E_h.
    int post_smoothing_iter = simpleSolver(H, W, d_divG, args[0], args+6, args[1], args[4], args[5], d_I_log);
    hipDeviceSynchronize();

    hipFree(d_r_h);
    hipFree(d_r_2h);
    hipFree(d_E_2h);
    hipFree(d_E_h);

    return pre_smoothing_iter + post_smoothing_iter + cycle_smoothing_iter;
}

int wCycleSolver(    
    const int H, const int W, 
    const float* d_divG, const float* args,
    float* d_I_log)
{
const int N = H * W;
    const int H2 = std::ceil(H / 2.0), W2 = std::ceil(W / 2.0);
    const int N2 = H2 * W2;

    dim3 nthreads_h(16, 16, 1);
    dim3 nblocks_h((W + nthreads_h.x - 1) / nthreads_h.x, (H + nthreads_h.y - 1) / nthreads_h.y, 1);

    dim3 nthreads_2h(16, 16, 1);
    dim3 nblocks_2h((W2 + nthreads_2h.x - 1) / nthreads_2h.x, (H2 + nthreads_2h.y - 1) / nthreads_2h.y, 1);
    
    // Step 1: Iterate on A_h * u = b_h to reach u_h (say 3 Jacobi or Gauss-Seidel steps)
    int pre_smoothing_iter = simpleSolver(H, W, d_divG, args[0], args+6, args[1], args[4], args[5], d_I_log);
    hipDeviceSynchronize();

    // Step 2: Restrict the residual r_h = b_h − A_h * u_h to the coarse grid by r_{2h} = R_{h}^{2h} * r_h
    float *d_r_h;
    hipMalloc(&d_r_h, N * sizeof(float));
    computeResidualKernel<<<nblocks_h, nthreads_h>>>(H, W, d_divG, d_I_log, d_r_h);
    hipDeviceSynchronize();

    float *d_r_2h;
    hipMalloc(&d_r_2h, N2 * sizeof(float));
    restrict2DKernel<<<nblocks_2h, nthreads_2h>>>(H, W, H2, W2, d_r_h, d_r_2h);
    hipDeviceSynchronize();

    // Step 3: Solve A_{2h} * E_{2h} = r_{2h} (or come close to E_{2h} by 3 iterations from E = 0)
    float *d_E_2h;
    hipMalloc(&d_E_2h, N2 * sizeof(float));
    hipMemset(d_E_2h, 0.0, N2 * sizeof(float));
    int cycle_smoothing_iter1;
    if (std::min(H2, W2) <= args[2])
        cycle_smoothing_iter1 = simpleSolver(H2, W2, d_r_2h, args[0], args+6, args[3], args[4], args[5], d_E_2h);
    else
        cycle_smoothing_iter1 = wCycleSolver(H2, W2, d_r_2h, args, d_E_2h);
    hipDeviceSynchronize();

    // Step 4: Interpolate E_{2h} back to E_h = I_{2h}^h * E_{2h}. Add E_h to u_h
    float *d_E_h;
    hipMalloc(&d_E_h, N * sizeof(float));
    interpolate2DKernel<<<nblocks_h, nthreads_h>>>(H, W, W2, d_E_2h, d_E_h);
    hipDeviceSynchronize();

    add2DKernel<<<nblocks_h, nthreads_h>>>(H, W, d_E_h, d_I_log);
    hipDeviceSynchronize();

    // Step 5: Iterate 3 more times on A_h * u = b_h starting from the improved u_h + E_h.
    int post_smoothing_iter1 = simpleSolver(H, W, d_divG, args[0], args+6, args[1], args[4], args[5], d_I_log);
    hipDeviceSynchronize();

    // Repeat from Step 2
    computeResidualKernel<<<nblocks_h, nthreads_h>>>(H, W, d_divG, d_I_log, d_r_h);
    hipDeviceSynchronize();

    restrict2DKernel<<<nblocks_2h, nthreads_2h>>>(H, W, H2, W2, d_r_h, d_r_2h);
    hipDeviceSynchronize();

    // Step 3
    hipMemset(d_E_2h, 0.0, N2 * sizeof(float));
    int cycle_smoothing_iter2;
    if (std::min(H2, W2) <= args[2])
        cycle_smoothing_iter2 = simpleSolver(H2, W2, d_r_2h, args[0], args+6, args[3], args[4], args[5], d_E_2h);
    else
        cycle_smoothing_iter2 = wCycleSolver(H2, W2, d_r_2h, args, d_E_2h);
    hipDeviceSynchronize();

    // Step 4
    interpolate2DKernel<<<nblocks_h, nthreads_h>>>(H, W, W2, d_E_2h, d_E_h);
    hipDeviceSynchronize();

    add2DKernel<<<nblocks_h, nthreads_h>>>(H, W, d_E_h, d_I_log);
    hipDeviceSynchronize();

    // Step 5
    int post_smoothing_iter2 = simpleSolver(H, W, d_divG, args[0], args+6, args[1], args[4], args[5], d_I_log);
    hipDeviceSynchronize();

    hipFree(d_r_h);
    hipFree(d_r_2h);
    hipFree(d_E_2h);
    hipFree(d_E_h);

    return pre_smoothing_iter + post_smoothing_iter1 + post_smoothing_iter2 + cycle_smoothing_iter1 + cycle_smoothing_iter2;
}

int fCycleSolver(    
    const int H, const int W, 
    const float* d_divG, const float* args,
    float* d_I_log)
{
    return -1;
}


int multigridSolver(
    const int H, const int W, 
    const float* d_divG, const int method, const float* args,
    const int iterations, const int checkFrequency, const float tolerance,
    float* d_I_log)
{
    const int N = H * W;

    mgMethodFunction mgMethodKernel;
    switch (method) {
        case 5:
            mgMethodKernel = vCycleSolver;
            break;
        case 6:
            mgMethodKernel = wCycleSolver;
            break;
        case 7:
            mgMethodKernel = fCycleSolver;
            break;
        default:
            return -1;
    }

    int nblocksError = (N + MAX_THREADS - 1) / MAX_THREADS;
    float h_error;
    float *d_error;
    hipMalloc(&d_error, sizeof(float));

    float *d_prev, *d_result;
    hipMalloc(&d_prev, N * sizeof(float));
    hipMemset(d_prev, 0.0, N * sizeof(float));
    hipMalloc(&d_result, N * sizeof(float));
    hipMemset(d_result, 0.0, N * sizeof(float));
    
    int total_iter_until_convergence = 0;
    for (int i = 0; i < iterations; ) 
    {
        if ((i+1) % checkFrequency == 0) hipMemcpy(d_prev, d_result, N * sizeof(float), hipMemcpyDeviceToDevice);
        total_iter_until_convergence += mgMethodKernel(H, W, d_divG, args, d_result);
        hipDeviceSynchronize(); 
        ++i;

        if (i % checkFrequency == 0)
        {   
            hipMemset(d_error, 0.0, sizeof(float));
            atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(d_result, d_prev, d_error, N);
            hipDeviceSynchronize();
            hipMemcpy(&h_error, d_error, sizeof(float), hipMemcpyDeviceToHost);
            h_error /= N;

            if (h_error < tolerance) break;
        }
    }

    hipMemcpy(d_I_log, d_result, N * sizeof(float), hipMemcpyDeviceToDevice);
    hipFree(d_result);
    hipFree(d_error);
    return total_iter_until_convergence;
}