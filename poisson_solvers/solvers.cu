#include "hip/hip_runtime.h"
#include "solvers.h"


/*
    Kernels for computing the residual/error between the previous and current iteration results.
*/
__global__ void blockErrorsKernel(const float* current, const float* previous, float* partialSums, const int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float s_sum[1024 / WARP_SIZE];

    int nwarps = blockDim.x / WARP_SIZE;
    int my_warp = threadIdx.x / WARP_SIZE;

    float sum = 0.0;
    
    if (tid < N)
        sum = fabsf(current[tid] - previous[tid]);
    __syncwarp();

    // shift within warp
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) 
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
    
    // sum over warps if needed
    if (nwarps > 1) {
        if (threadIdx.x % WARP_SIZE == 0)
            s_sum[my_warp] = sum;
        __syncthreads();

        if (threadIdx.x == 0) {
            for (int i = 1; i < nwarps; ++i)
                sum += s_sum[i];
        }
    }

    // final step - store results into the main device memory
    if (threadIdx.x == 0) {
        partialSums[blockIdx.x] = sum;
    }
}

__global__ void blockErrorsReductionKernel(const float* partialSums, float* result, const int numSums) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float s_sum[1024 / WARP_SIZE];
    
    int nwarps = blockDim.x / WARP_SIZE;
    int my_warp = threadIdx.x / WARP_SIZE;

    float sum = 0.0;
    
    if (tid < numSums)
        sum = partialSums[tid];
    __syncwarp();

    // shift within warp
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) 
        sum += __shfl_down_sync(FULL_MASK, sum, offset);

    // sum over warps if needed
    if (nwarps > 1) {
        if (threadIdx.x % WARP_SIZE == 0)
            s_sum[my_warp] = sum;
        __syncthreads();

        if (threadIdx.x == 0) {
            for (int i = 1; i < nwarps; ++i)
                sum += s_sum[i];
        }
    }

    // final step - store results into the main device memory
    if (threadIdx.x == 0) {
        result[blockIdx.x] = sum;
    }
}

__global__ void atomicAddBlockErrorsKernel(const float* current, const float* previous, float* result, const int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float s_sum[1024 / WARP_SIZE];

    int nwarps = blockDim.x / WARP_SIZE;
    int my_warp = threadIdx.x / WARP_SIZE;

    float sum = 0.0;
    
    if (tid < N)
        sum = fabsf(current[tid] - previous[tid]);
    __syncwarp();

    // shift within warp
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) 
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
    
    // sum over warps if needed
    if (nwarps > 1) {
        if (threadIdx.x % WARP_SIZE == 0)
            s_sum[my_warp] = sum;
        __syncthreads();

        if (threadIdx.x == 0) {
            for (int i = 1; i < nwarps; ++i)
                sum += s_sum[i];
        }
    }

    // final step - store results into the main device memory
    if (threadIdx.x == 0) {
        atomicAdd(result, sum);
    }
}

/*
    Kernel for the Jacobi method.
*/
__global__ void jacobiKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    float up = (j > 0) ? current[idx - W] : current[idx];
    float left = (i > 0) ? current[idx - 1] : current[idx];
    float right = (i < W - 1) ? current[idx + 1] : current[idx];
    float bottom = (j < H - 1) ? current[idx + W] : current[idx];

    result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
}

void jacobi(const int H, const int W, const float* d_divG, const float* d_current, const dim3 nblocks, const dim3 nthreads, float* d_I_log)
{
    jacobiKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_I_log);
}


/*
    Kernels for the Red-Black Gauss-Seildel method.
*/
__global__ void redGaussSeidelKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float up = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel(const int H, const int W, const float* b, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float up = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];
        result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
}

void gaussSeidelRedBlack(const int H, const int W, const float* d_divG, const float* d_current, const dim3 nblocks, const dim3 nthreads, float* d_I_log)
{
    redGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_I_log);
    blackGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_I_log);
}


/*
    Kernels for the Red-Black Gauss-Seildel method with overrelaxation.
*/
__global__ void redGaussSeidelSORKernel(const int H, const int W, const float* b, const float *current, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float up = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        
        float GS_update = 0.25f * (up + left + right + bottom - b[idx]);
        result[idx] = (1 - w_opt) * current[idx] + w_opt * GS_update;
    }
}

__global__ void blackGaussSeidelSORKernel(const int H, const int W, const float* b, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float up = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];

        float GS_update = 0.25f * (up + left + right + bottom - b[idx]);
        result[idx] = (1 - w_opt) * result[idx] + w_opt * GS_update;
    }
}

void gaussSeidelRedBlackSOR(const int H, const int W, const float* d_divG, const float* d_current, const dim3 nblocks, const dim3 nthreads, float* d_I_log)
{
    float w_opt = 2.0f / (1.0f + sqrtf(1.0f - powf(cosf(M_PI / max(H, W)), 2.0f)));
    redGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_I_log, w_opt);
    blackGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_I_log, w_opt);
}


int solver(
    const int H, const int W, 
    const float* d_divG, const int method,
    const int iterations, const float tolerance, const int checkFrequency,
    float* d_I_log)
{
    const int N = H * W;

    dim3 nthreadsMethod(16, 16, 1);
    dim3 nblocksMethod((W + nthreadsMethod.x - 1) / nthreadsMethod.x, (H + nthreadsMethod.y - 1) / nthreadsMethod.y, 1);

    float *d_current;
    hipMalloc(&d_current, N * sizeof(float));
    hipMemset(d_current, 0.0, N * sizeof(float));
    hipMemset(d_I_log, 0.0, N * sizeof(float));

    methodFunction methodKernel;
    switch (method) {
        case 0:
            methodKernel = jacobi;
            break;
        case 1:
            methodKernel = gaussSeidelRedBlack;
            break;
        case 2:
            methodKernel = gaussSeidelRedBlackSOR;
            break;
        default:
            return -1;
    }

    int i = 0;
    if (tolerance < 0.0)
    { 
        for (; i < iterations; ) 
        {
            methodKernel(H, W, d_divG, d_current, nblocksMethod, nthreadsMethod, d_I_log);
            hipDeviceSynchronize();
            std::swap(d_current, d_I_log); ++i;
        }
    } 
    else 
    {   
        int nblocksError = (N + MAX_THREADS - 1) / MAX_THREADS;
        float *partialErrorSums;
        hipMalloc(&partialErrorSums, nblocksError * sizeof(float));
        hipMemset(partialErrorSums, 0.0, nblocksError * sizeof(float));

        int nblocksError2 = (nblocksError + MAX_THREADS - 1) / MAX_THREADS;
        float* partialErrorSums2;
        hipMalloc(&partialErrorSums2, nblocksError2 * sizeof(float)); // This should be enough for 2^30 elements/pixels
        hipMemset(partialErrorSums2, 0.0, nblocksError2 * sizeof(float));

        float error_h;
        float *error_d;
        hipMalloc(&error_d, sizeof(float));

        for (; i < iterations; ) 
        {
            methodKernel(H, W, d_divG, d_current, nblocksMethod, nthreadsMethod, d_I_log);
            hipDeviceSynchronize();
            std::swap(d_current, d_I_log); ++i;

            if (i % checkFrequency == 0) // This error calculation may be inefficient. Possibly be improved in the future.
            {   
                blockErrorsKernel<<<nblocksError, MAX_THREADS>>>(d_current, d_I_log, partialErrorSums, N);
                if (nblocksError > MAX_THREADS)
                {
                    blockErrorsReductionKernel<<<nblocksError2, MAX_THREADS>>>(partialErrorSums, partialErrorSums2, nblocksError);
                    blockErrorsReductionKernel<<<1, MAX_THREADS>>>(partialErrorSums2, error_d, nblocksError2);
                }
                else
                {
                    blockErrorsReductionKernel<<<1, MAX_THREADS>>>(partialErrorSums, error_d, nblocksError);
                }
                // atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(d_current, d_I_log, error_d, N);
                hipDeviceSynchronize();
                hipMemcpy(&error_h, error_d, sizeof(float), hipMemcpyDeviceToHost);
                
                error_h /= N;

                if (error_h < tolerance) break;
            }
        }
        hipFree(partialErrorSums);
        hipFree(partialErrorSums2);
        hipFree(error_d);
    }

    if (i % 2 == 1) std::swap(d_current, d_I_log);
    hipFree(d_current);
    return i;
}


void fullMultigridSolver(    
    const int H, const int W, 
    const float* d_divG, 
    const int iterations, const float tolerence,
    float* d_I_log)
{

}
