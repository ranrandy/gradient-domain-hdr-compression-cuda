#include "hip/hip_runtime.h"
#include "solvers.h"


/*
    Computing the error between the previous and current iteration results. In most cases, atomicAdd should be fast enough.
*/
__global__ void atomicAddBlockErrorsKernel(const float* current, const float* previous, float* result, const int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float s_sum[1024 / WARP_SIZE];

    int nwarps = blockDim.x / WARP_SIZE;
    int my_warp = threadIdx.x / WARP_SIZE;

    float sum = 0.0;
    
    if (tid < N)
        sum = fabsf(current[tid] - previous[tid]);
    __syncwarp();

    // shift within warp
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) 
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
    
    // sum over warps if needed
    if (nwarps > 1) {
        if (threadIdx.x % WARP_SIZE == 0)
            s_sum[my_warp] = sum;
        __syncthreads();

        if (threadIdx.x == 0) {
            for (int i = 1; i < nwarps; ++i)
                sum += s_sum[i];
        }
    }

    // final step - store results into the main device memory
    if (threadIdx.x == 0) {
        atomicAdd(result, sum);
    }
}

/*
    0. Jacobi method.
*/
__global__ void jacobiKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    float up = (j > 0) ? current[idx - W] : current[idx];
    float left = (i > 0) ? current[idx - 1] : current[idx];
    float right = (i < W - 1) ? current[idx + 1] : current[idx];
    float bottom = (j < H - 1) ? current[idx + W] : current[idx];

    result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
}

void jacobi(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_result, const float* args)
{
    jacobiKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result);
}


/*
    1. Red-Black Gauss-Seildel method.
*/
__global__ void redGaussSeidelKernel(const int H, const int W, const float* b, const float *current, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float up = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel(const int H, const int W, const float* b, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float up = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];
        result[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
}

void gaussSeidelRedBlack(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_result, const float* args)
{
    redGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result);
    blackGaussSeidelKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_result);
}


/*
    2. Red-Black Gauss-Seildel method with successive over relaxation (SOR).
*/
__global__ void redGaussSeidelSORKernel(const int H, const int W, const float* b, const float *current, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 0)
    {
        float up = (j > 0) ? current[idx - W] : current[idx];
        float left = (i > 0) ? current[idx - 1] : current[idx];
        float right = (i < W - 1) ? current[idx + 1] : current[idx];
        float bottom = (j < H - 1) ? current[idx + W] : current[idx];
        result[idx] = (1 - w_opt) * current[idx] + w_opt * 0.25f * (up + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelSORKernel(const int H, const int W, const float* b, const float *current, float *result, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if ((i + j) % 2 == 1)
    {
        float up = (j > 0) ? result[idx - W] : result[idx];
        float left = (i > 0) ? result[idx - 1] : result[idx];
        float right = (i < W - 1) ? result[idx + 1] : result[idx];
        float bottom = (j < H - 1) ? result[idx + W] : result[idx];
        result[idx] = (1 - w_opt) * current[idx] + w_opt * 0.25f * (up + left + right + bottom - b[idx]);
    }
}

void gaussSeidelRedBlackSOR(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* d_current, float* d_result, const float* args)
{
    // float w_opt = 2.0f / (1.0f + sinf(M_PI / max(H, W)));
    redGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result, args[0]);
    blackGaussSeidelSORKernel<<<nblocks, nthreads>>>(H, W, d_divG, d_current, d_result, args[0]);
}


/*
    --> Red-Black pre-reordering of grids.
*/
__global__ void fillInRedBlackInitGaussSeidel2Kernel(const int H, const int W, const float* init_guess, float* red, float* black) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;
    int squeezed_j;

    if ((i+j) % 2 == 0) // red
    {
        if (i % 2 == 1) squeezed_j = (j - 1) / 2;
        else squeezed_j = j / 2;
        red[squeezed_j * W + i] = init_guess[idx];
    }
    else // black
    {
        if (i % 2 == 1) squeezed_j = j / 2;
        else squeezed_j = (j - 1) / 2; 
        black[squeezed_j * W + i] = init_guess[idx];
    }
}

__global__ void fillInRedBlackBGaussSeidel2Kernel(const int H, const int W, const float* b, float* red_black_b, const int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;
    int squeezed_j;

    if ((i+j) % 2 == 0) // red
    {
        if (i % 2 == 1) squeezed_j = (j - 1) / 2;
        else squeezed_j = j / 2;
        red_black_b[squeezed_j * W + i] = b[idx];
    }
    else // black
    {
        if (i % 2 == 1) squeezed_j = j / 2;
        else squeezed_j = (j - 1) / 2; 
        red_black_b[squeezed_j * W + i + offset] = b[idx];
    }
}

__global__ void fillInGaussSeidel2Kernel(const int H, const int W, const float* red, const float* black, float* result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;
    int squeezed_j;

    if ((j + i) % 2 == 0) // red
    {
        if (i % 2 == 1) squeezed_j = (j - 1) / 2;
        else squeezed_j = j / 2;
        int squeezed_idx = squeezed_j * W + i; 
        result[idx] = red[squeezed_idx];
    }
    else // black
    {
        if (i % 2 == 1) squeezed_j = j / 2;
        else squeezed_j = (j - 1) / 2; 
        int squeezed_idx = squeezed_j * W + i; 
        result[idx] = black[squeezed_idx];
    }
}

/*
    3. Gauss-Seidel with pre-reordering of grids.
*/
__global__ void redGaussSeidelKernel2(const int H, const int W, const float* b, const float *black, float *red) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float up = black[idx];
        float left = black[idx - 1];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx + W] : red[idx];
        red[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
    else
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float up = (orig_j > 0) ? black[idx - W] : red[idx];
        float left = (i > 0) ? black[idx - 1] : red[idx];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx] : red[idx];
        red[idx] = 0.25f * (up + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel2(const int H, const int W, const float* b, const float *red, float *black, const int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float up = (orig_j > 0) ? red[idx - W] : black[idx];
        float left = red[idx - 1];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx] : black[idx];
        black[idx] = 0.25f * (up + left + right + bottom - b[idx + offset]);
    }
    else
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float up = red[idx];
        float left = (i > 0) ? red[idx - 1] : black[idx];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx + W] : black[idx];
        black[idx] = 0.25f * (up + left + right + bottom - b[idx + offset]);
    }
}

void gaussSeidelRedBlack2(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* red, float* black, const float* args)
{
    redGaussSeidelKernel2<<<nblocks, nthreads>>>(H, W, d_divG, black, red);
    blackGaussSeidelKernel2<<<nblocks, nthreads>>>(H, W, d_divG, red, black, (H * W) / 2);
}

/*
    4. Gauss-Seidel with SOR and pre-reordering of grids.
*/
__global__ void redGaussSeidelKernel2SOR(const int H, const int W, const float* b, const float *black, float *red, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float up = black[idx];
        float left = black[idx - 1];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx + W] : red[idx];
        red[idx] = (1 - w_opt) * red[idx] + w_opt * 0.25f * (up + left + right + bottom - b[idx]);
    }
    else
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float up = (orig_j > 0) ? black[idx - W] : red[idx];
        float left = (i > 0) ? black[idx - 1] : red[idx];
        float right = (i < W - 1) ? black[idx + 1] : red[idx];
        float bottom = (orig_j < H - 1) ? black[idx] : red[idx];
        red[idx] = (1 - w_opt) * red[idx] + w_opt * 0.25f * (up + left + right + bottom - b[idx]);
    }
}

__global__ void blackGaussSeidelKernel2SOR(const int H, const int W, const float* b, const float *red, float *black, const int offset, const float w_opt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W) return;

    int idx = j * W + i;

    if (i % 2 == 1)
    {
        int orig_j = j * 2;
        if (orig_j >= H) return;

        float up = (orig_j > 0) ? red[idx - W] : black[idx];
        float left = red[idx - 1];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx] : black[idx];
        black[idx] = (1 - w_opt) * black[idx] + w_opt * 0.25f * (up + left + right + bottom - b[idx + offset]);
    }
    else
    {
        int orig_j = j * 2 + 1;
        if (orig_j >= H) return;

        float up = red[idx];
        float left = (i > 0) ? red[idx - 1] : black[idx];
        float right = (i < W - 1) ? red[idx + 1] : black[idx];
        float bottom = (orig_j < H - 1) ? red[idx + W] : black[idx];
        black[idx] = (1 - w_opt) * black[idx] + w_opt * 0.25f * (up + left + right + bottom - b[idx + offset]);
    }
}

void gaussSeidelRedBlack2SOR(const int H, const int W, const float* d_divG, const dim3 nblocks, const dim3 nthreads, float* red, float* black, const float* args)
{
    redGaussSeidelKernel2SOR<<<nblocks, nthreads>>>(H, W, d_divG, black, red, args[0]);
    blackGaussSeidelKernel2SOR<<<nblocks, nthreads>>>(H, W, d_divG, red, black, (H * W) / 2, args[0]);
}


int simpleSolver(
    const int H, const int W, 
    const float* d_divG, const int method, const float* args, const float* d_init_guess, 
    const int iterations, const int checkFrequency, const float tolerance,
    float* d_I_log)
{
    const int N = H * W, N2 = H * W / 2;

    methodFunction methodKernel;
    switch (method) {
        case 0:
            methodKernel = jacobi;
            break;
        case 1:
            methodKernel = gaussSeidelRedBlack;
            break;
        case 2:
            methodKernel = gaussSeidelRedBlackSOR;
            break;
        case 3:
            methodKernel = gaussSeidelRedBlack2;
            break;
        case 4:
            methodKernel = gaussSeidelRedBlack2SOR;
            break;
        default:
            return -1;
    }

    int nblocksError = (N + MAX_THREADS - 1) / MAX_THREADS;
    float h_error;
    float *d_error;
    hipMalloc(&d_error, sizeof(float));

    if (method <= 2) 
    {                
        dim3 nthreadsMethod(16, 16, 1);
        dim3 nblocksMethod((W + nthreadsMethod.x - 1) / nthreadsMethod.x, (H + nthreadsMethod.y - 1) / nthreadsMethod.y, 1);

        float *d_current, *d_result;
        hipMalloc(&d_current, N * sizeof(float));
        hipMemset(d_current, 0.0, N * sizeof(float));
        if (d_init_guess != nullptr) hipMemcpy(d_current, d_init_guess, N * sizeof(float), hipMemcpyDeviceToDevice);

        hipMalloc(&d_result, N * sizeof(float));
        hipMemset(d_result, 0.0, N * sizeof(float));

        int i = 0;
        for (; i < iterations; ) 
        {
            methodKernel(H, W, d_divG, nblocksMethod, nthreadsMethod, d_current, d_result, args);
            hipDeviceSynchronize(); 
            std::swap(d_current, d_result); ++i;

            if (i % checkFrequency == 0)
            {   
                hipMemset(d_error, 0.0, sizeof(float));
                atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(d_current, d_result, d_error, N);
                hipDeviceSynchronize();
                hipMemcpy(&h_error, d_error, sizeof(float), hipMemcpyDeviceToHost);
                h_error /= N;

                if (h_error < tolerance) break;
            }
        }

        if (i % 2 == 1) std::swap(d_current, d_result);
        hipMemcpy(d_I_log, d_result, N * sizeof(float), hipMemcpyDeviceToDevice);
        hipFree(d_current);
        hipFree(d_result);
        hipFree(d_error);
        return i;
    }

    if (method > 2) 
    {
        dim3 nthreadsMethod(32, 16, 1);
        dim3 nblocksMethod((W + nthreadsMethod.x - 1) / nthreadsMethod.x, (int(H/2) + nthreadsMethod.y - 1) / nthreadsMethod.y, 1);

        dim3 nthreadsFillIn(16, 16, 1);
        dim3 nblocksFillIn((W + nthreadsFillIn.x - 1) / nthreadsFillIn.x, (H + nthreadsFillIn.y - 1) / nthreadsFillIn.y, 1);

        float *red, *black;
        hipMalloc(&red, N2 * sizeof(float));
        hipMemset(red, 0.0, N2 * sizeof(float));
        hipMalloc(&black, N2 * sizeof(float));
        hipMemset(black, 0.0, N2 * sizeof(float));
        if (d_init_guess != nullptr) fillInRedBlackInitGaussSeidel2Kernel<<<nblocksFillIn, nthreadsFillIn>>>(H, W, d_init_guess, red, black);

        float *prev_red;
        hipMalloc(&prev_red, N2 * sizeof(float));

        float *red_black_divG;
        hipMalloc(&red_black_divG, N * sizeof(float));
        fillInRedBlackBGaussSeidel2Kernel<<<nblocksFillIn, nthreadsFillIn>>>(H, W, d_divG, red_black_divG, N2);

        int i = 0;
        for (; i < iterations; ) 
        {
            if ((i+1) % checkFrequency == 0) hipMemcpy(prev_red, red, N2 * sizeof(float), hipMemcpyDeviceToDevice);

            methodKernel(H, W, red_black_divG, nblocksMethod, nthreadsMethod, red, black, args);
            hipDeviceSynchronize();
            ++i;

            if (i % checkFrequency == 0)
            {   
                hipMemset(d_error, 0.0, sizeof(float));
                atomicAddBlockErrorsKernel<<<nblocksError, MAX_THREADS>>>(red, prev_red, d_error, N2);
                hipDeviceSynchronize();
                hipMemcpy(&h_error, d_error, sizeof(float), hipMemcpyDeviceToHost);
                h_error /= N2;

                if (h_error < tolerance) break;
            }
        }

        fillInGaussSeidel2Kernel<<<nblocksFillIn, nthreadsFillIn>>>(H, W, red, black, d_I_log);
        hipDeviceSynchronize();

        hipFree(red);
        hipFree(black);
        hipFree(prev_red);
        hipFree(red_black_divG);
        hipFree(d_error);
        return i;
    }

    hipFree(d_error);
    return -1;
}


__global__ void computeResidualKernel(const int H, const int W, const float* b_h, const float* u_h, float* r_h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= W || j >= H) return;

    int idx = j * W + i;

    float up = (j > 0) ? u_h[idx - W] : u_h[idx];
    float left = (i > 0) ? u_h[idx - 1] : u_h[idx];
    float right = (i < W - 1) ? u_h[idx + 1] : u_h[idx];
    float bottom = (j < H - 1) ? u_h[idx + W] : u_h[idx];

    r_h[idx] = b_h[idx] - (up + left + right + bottom - 4 * u_h[idx]);
}

__global__ void restrict1DKernel(const int N, const int N2, const float* r_h, float* r_2h) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N2) return;

    int idx = tid * 2 + 1;

    float up = r_h[idx-1];
    float middle = r_h[idx];
    float bottom = (idx < N - 1) ? r_h[idx+1] : r_h[idx];

    r_2h[tid] = 0.25 * (up + middle * 2 + bottom);
}

__global__ void interpolate1DKernel(const int N, const float* E_2h, float* E_h) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = tid / 2;

    if (tid >= N) return;

    float left, right;

    if (tid % 2 == 0) 
    {
        left = (idx > 0) ? E_2h[idx-1] : 0;
        right = E_2h[idx];
    }
    else
    {
        left = E_2h[idx];
        right = E_2h[idx];
    }

    E_h[tid] = 0.5 * (left + right);
}

__global__ void add1DKernel(const int N, const float* E_h, float* u_h) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    u_h[tid] += E_h[tid];
}


int multigridSolver(    
    const int H, const int W, 
    const float* d_divG, const int method, const float* args,
    const int iterations, const int checkFrequency, const float tolerance,
    float* d_I_log)
{
    const int N = H * W, N2 = H * W / 2;
    const int H2 = H / 2, W2 = W / 2;

    // Step 1: Iterate on A_h * u = b_h to reach u_h (say 3 Jacobi or Gauss-Seidel steps)
    float *d_u_h;
    hipMalloc(&d_u_h, N * sizeof(float));
    int pre_smoothing_iter = simpleSolver(H, W, d_divG, args[0], args+6, nullptr, args[1], args[4], args[5], d_u_h);
    hipDeviceSynchronize();

    // Step 2: Restrict the residual r_h = b_h − A_h * u_h to the coarse grid by r_{2h} = R_{h}^{2h} * r_h
    float *d_r_h;
    hipMalloc(&d_r_h, N * sizeof(float));
    dim3 nthreadsResidual(16, 16, 1);
    dim3 nblocksResidual((W + nthreadsResidual.x - 1) / nthreadsResidual.x, (H + nthreadsResidual.y - 1) / nthreadsResidual.y, 1);
    computeResidualKernel<<<nblocksResidual, nthreadsResidual>>>(H, W, d_divG, d_u_h, d_r_h);
    hipDeviceSynchronize();

    float *d_r_2h;
    hipMalloc(&d_r_2h, N2 * sizeof(float));
    dim3 nthreadsRestrict1D(256, 1, 1);
    dim3 nblocksRestrict1D((N2 + nthreadsRestrict1D.x - 1) / nthreadsRestrict1D.x, 1, 1);
    restrict1DKernel<<<nblocksRestrict1D, nthreadsRestrict1D>>>(N, N2, d_r_h, d_r_2h);
    hipDeviceSynchronize();

    // Step 3: Solve A_{2h} * E_{2h} = r_{2h} (or come close to E_{2h} by 3 iterations from E = 0)
    float *d_E_2h;
    hipMalloc(&d_E_2h, N2 * sizeof(float));
    int cycle_smoothing_iter = simpleSolver(H2, W2, d_r_2h, args[0], args+6, nullptr, args[3], args[4], args[5], d_E_2h);
    hipDeviceSynchronize();

    // Step 4: Interpolate E_{2h} back to E_h = I_{2h}^h * E_{2h}. Add E_h to u_h
    float *d_E_h;
    hipMalloc(&d_E_h, N * sizeof(float));
    dim3 nthreadsInterpolate1D(256, 1, 1);
    dim3 nblocksInterpolate1D((N + nthreadsInterpolate1D.x - 1) / nthreadsInterpolate1D.x, 1, 1);
    interpolate1DKernel<<<nblocksInterpolate1D, nthreadsInterpolate1D>>>(N, d_E_2h, d_E_h);
    hipDeviceSynchronize();

    // Step 5: Iterate 3 more times on A_h * u = b_h starting from the improved u_h + E_h.
    dim3 nthreadsAdd1D(256, 1, 1);
    dim3 nblocksAdd1D((N + nthreadsAdd1D.x - 1) / nthreadsAdd1D.x, 1, 1);
    add1DKernel<<<nblocksAdd1D, nthreadsAdd1D>>>(N, d_E_h, d_u_h);
    hipDeviceSynchronize();

    int post_smoothing_iter = simpleSolver(H, W, d_divG, args[0], args+6, d_u_h, args[1], args[4], args[5], d_I_log);
    hipDeviceSynchronize();

    hipFree(d_u_h);
    hipFree(d_r_h);
    hipFree(d_r_2h);
    hipFree(d_E_2h);
    hipFree(d_E_h);

    return pre_smoothing_iter + post_smoothing_iter + cycle_smoothing_iter;
}
